#include "db_async_cuda.h"
#include <iostream>
#include <ATen/cuda/HIPContext.h>


#include "../thread.h"
#include "../gasdb.h"

extern GasDb * gasdb;


Thread &getThread() {
  static Thread thread;
  return thread;
}


void db_synchronize_cuda() {
  std::cout << "syncing thread\n";
  getThread().synchronize();
}

void db_read_async_cuda(int64_t layer, torch::optional<torch::Tensor> optional_offset,
                     torch::optional<torch::Tensor> optional_count,
                     torch::Tensor index, torch::Tensor dst) {


  std::cout << "db_read_async_cuda at layer " << layer << std::endl;
  AT_ASSERTM(!index.is_cuda(), "Index tensor must be a CPU tensor");
  AT_ASSERTM(dst.is_cuda(), "Target tensor must be a CUDA tensor");

  AT_ASSERTM(dst.is_contiguous(), "Target tensor must be contiguous");

  AT_ASSERTM(index.dim() == 1, "Index tensor must be one-dimensional");

  int64_t numel = 0;
  if (optional_offset.has_value()) {
    auto offset = optional_offset.value();
    AT_ASSERTM(!offset.is_cuda(), "Offset tensor must be a CPU tensor");
    AT_ASSERTM(offset.is_contiguous(), "Offset tensor must be contiguous");
    AT_ASSERTM(offset.dim() == 1, "Offset tensor must be one-dimensional");
    AT_ASSERTM(optional_count.has_value(), "Count tensor is undefined");
    auto count = optional_count.value();
    AT_ASSERTM(!count.is_cuda(), "Count tensor must be a CPU tensor");
    AT_ASSERTM(count.is_contiguous(), "Count tensor must be contiguous");
    AT_ASSERTM(count.dim() == 1, "Count tensor must be one-dimensional");
    AT_ASSERTM(offset.numel() == count.numel(), "Size mismatch");
    numel = count.sum().data_ptr<int64_t>()[0];
  }

  AT_ASSERTM(numel + index.numel() <= dst.size(0),
             "Target tensor size too small");

  auto stream = at::cuda::getCurrentCUDAStream(dst.get_device());
  AT_ASSERTM(stream != at::cuda::getDefaultCUDAStream(dst.get_device()),
             "Asynchronous read requires a non-default CUDA stream");

  std::cout << "queuing new task in thread to populate src tensor then lauch hipMemcpyAsync\n";
  AT_DISPATCH_ALL_TYPES(dst.scalar_type(), "db_read_async", [&] {
    getThread().run([=] {
      std::cout << "lauching populate src tensor then lauch hipMemcpyAsync\n";
      auto options = torch::TensorOptions().dtype(torch::kFloat32);
      auto dst_data = dst.data_ptr<scalar_t>();

      int64_t numel_offset = 0;
      if (optional_offset.has_value())
        numel_offset = torch::sum(optional_count.value()).item<int64_t>();

      int64_t numel = numel_offset  + index.numel();
      int64_t esize = dst.numel()/dst.size(0);

      torch::Tensor src=torch::empty({numel, esize},options);
      auto src_data = src.data_ptr<scalar_t>();

      if (optional_offset.has_value()) {
        auto offset = optional_offset.value();
        auto count = optional_count.value();
        auto offset_data = offset.data_ptr<int64_t>();
        auto count_data = count.data_ptr<int64_t>();

        for (int64_t i = 0; i < offset.numel(); i++)
          for (int64_t j = 0; j < count_data[i]; j++)
            src.index_put_({offset_data[i]+j},gasdb->pull(offset_data[i]+j, layer));
      }
      for (int64_t i = 0; i < index.numel(); i++)
        src.index_put_({numel_offset+i},gasdb->pull(index[i].item<int64_t>(), layer));

      hipMemcpyAsync(dst_data, src_data,
                      src.numel() * esize * sizeof(scalar_t),
                      hipMemcpyHostToDevice, stream);

    });
  });
}

void db_write_async_cuda(int64_t layer, torch::Tensor src, torch::Tensor offset,
                      torch::Tensor count) {
  std::cout << "db_write_async_cuda at layer " << layer << std::endl;
  AT_ASSERTM(src.is_cuda(), "Source tensor must be a CUDA tensor");
  AT_ASSERTM(!offset.is_cuda(), "Offset tensor must be a CPU tensor");
  AT_ASSERTM(!count.is_cuda(), "Count tensor must be a CPU tensor");

  AT_ASSERTM(src.is_contiguous(), "Index tensor must be contiguous");
  AT_ASSERTM(offset.is_contiguous(), "Offset tensor must be contiguous");
  AT_ASSERTM(count.is_contiguous(), "Count tensor must be contiguous");

  AT_ASSERTM(offset.dim() == 1, "Offset tensor must be one-dimensional");
  AT_ASSERTM(count.dim() == 1, "Count tensor must be one-dimensional");
  AT_ASSERTM(offset.numel() == count.numel(), "Size mismatch");

  std::cout << "queuing new task in thread to  get cuda tensor then push to db\n";
  AT_DISPATCH_ALL_TYPES(src.scalar_type(), "db_write_async", [&] {
    getThread().run([=] {
      std::cout << "lauching  get cuda tensor then push to db\n";

      auto options = torch::TensorOptions().dtype(torch::kFloat32);
      int64_t numel = torch::sum(count).item<int64_t>();
      int64_t esize = src.numel()/src.size(0);
      torch::Tensor dst=torch::empty({numel, esize},options);

      auto dst_data = dst.data_ptr<scalar_t>();
      auto src_data = src.data_ptr<scalar_t>();

      hipMemcpy(dst_data, src_data, numel*esize*sizeof(scalar_t), hipMemcpyDeviceToHost);

      gasdb->push(dst, layer, offset, count);
    });
  });
}
